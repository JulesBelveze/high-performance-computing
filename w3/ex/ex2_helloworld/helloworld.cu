#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>



void timestamp()
{
    time_t ltime; /* calendar time */
    ltime=time(NULL); /* get current cal time */
    printf("%s",asctime( localtime(&ltime) ) );
}

__global__ void use_local_memory()
{
  int g_tid = threadIdx.x + blockIdx.x * blockDim.x;
  // printf("Global Thread ID: %d\n",g_tid);
  int b_tid = threadIdx.x;
  // printf("Block Thread ID: %d\n",b_tid);
  int bid = blockIdx.x;
  // printf("Block ID: %d\n",bid);
  printf("Hello world! I’m thread %d out of 64 in block %d. My global thread id is %d out of 256.\n",b_tid,bid,g_tid);
}

#define N 256
int main() {
  printf("Hello World!\n");
  timestamp();
  // Launch kernel using 30 threads per block
  use_local_memory<<<N/30, 30>>>();
  hipDeviceSynchronize();
}
