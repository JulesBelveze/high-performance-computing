#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

void timestamp()
{
    time_t ltime; /* calendar time */
    ltime=time(NULL); /* get current cal time */
    printf("%s",asctime( localtime(&ltime) ) );
}

__global__ void hello_thread()
{
  int g_tid = threadIdx.x + blockIdx.x * blockDim.x;
  int b_tid = threadIdx.x;
  int bid = blockIdx.x;
  // 258 without if statement
  // 194 with if statement
  // 64 lines missing
  if(g_tid == 100){
     int *a = (int*) 0x10000; *a = 0;
  }
  printf("Hello world! I’m thread %d out of %d in block %d. My global thread id is %d out of %d.\n",b_tid,blockDim.x,bid,g_tid,gridDim.x*blockDim.x);
}

#define total_threads 256
#define threads_per_block 64
int main() {
  printf("Hello World!\n");
  timestamp();
  // Launch kernel using 64 threads per block
  hello_thread<<<total_threads/threads_per_block, threads_per_block>>>();
  // hipDeviceSynchronize();
  checkCudaErrors(hipDeviceSynchronize());
}
