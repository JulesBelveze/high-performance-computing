// GPU Sequential version v1

void gpu1(int m, int n, int k, double *a, double *b, double *c){
    int i, j, p;

    for(i = 0; i < m*n; i++){
        c[i] = 0.0;
    }

    for(i = 0; i < m; i++){
        for(p = 0; p < k; p++){
            for(j = 0; j < m; j++){
                c[i*n+j] += a[i*k+p]*b[p*n+j];
            }
        }
    }
}


void matmult_gpu1(int m, int n, int k,  double *a, double *b, double *c){
    double *d_a, *d_b, *d_c;

    // allocate space for device copies
    hipMalloc((void **)&d_a, n*k*sizeof(double));
    hipMalloc((void **)&d_b, k*m*sizeof(double));
    hipMalloc((void **)&d_c, m*n*sizeof(double));

    // copying input to device
    hipMemcpy(d_a, a, n*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k*m*sizeof(double), hipMemcpyHostToDevice);

    gpu1<<<1,1>>>(m, n, k, d_a, d_b, d_c);

    hipDeviceSynchronize();

    // copying output to host
    hipMemcpy(c, d_c, n*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}
