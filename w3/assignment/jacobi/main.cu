#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <omp.h>
#include <string.h>

#include "datatools.h"
#include "jacobi_library.h"

int main(int argc, char *argv[])
{

    int n, num_iterations;
    double ts, te;
    double *u, *u_old, *f, *u_new;
    char type[4];

    if (argc == 2)
    {
        strcpy(type, argv[1]);
        n = 100;
        num_iterations = 10000;
    }
    else if (argc == 3)
    {
        strcpy(type, argv[1]);
        n = atoi(argv[2]);
        num_iterations = 10000;
    }
    else if (argc == 4)
    {
        strcpy(type, argv[1]);
        n = atoi(argv[2]);
        num_iterations = atoi(argv[3]);
    }
    else
    {
        strcpy(type, "cpu");
        n = 100;
        num_iterations = 10000;
    }

    /* Allocate memory*/
    u     = (double *)malloc((n + 2) * (n + 2) * sizeof(double));
    u_old = (double *)malloc((n + 2) * (n + 2) * sizeof(double));
    u_new = (double *)malloc((n + 2) * (n + 2) * sizeof(double));
    f     = (double *)malloc((n + 2) * (n + 2) * sizeof(double));

    if (u == NULL || u_old == NULL | f == NULL | u_new == NULL)
    {
        fprintf("Memory allocation error...\n");
        exit(EXIT_FAILURE);
    }

    /* Initialize data */
    init_data(n, u, f);

    /* Call the function */
    if (strcmp(type, "cpu") == 0)
    {
        ts = omp_get_wtime();
        jacobi_cpu(n, num_iterations, f, u);
        te = omp_get_wtime() - ts;
    }
    else if (strcmp(type, "gpu1") == 0)
    {
        ts = omp_get_wtime();
        jacobi_gpu1(n, num_iterations, f, u);
        te = omp_get_wtime() - ts;
    }
    else if (strcmp(type, "gpu2") == 0)
    {
        ts = omp_get_wtime();
        jacobi_gpu2(n, num_iterations, f, u);
        te = omp_get_wtime() - ts;
    }
    else if (strcmp(type, "gpu3") == 0)
    {
        ts = omp_get_wtime();
        jacobi_gpu3(n, num_iterations, f, u);
        te = omp_get_wtime() - ts;
    }

    /* Free memory */
    hipFree(u); hipFree(u_old); hipFree(u_new); hipFree(f);

    return (0);
}
