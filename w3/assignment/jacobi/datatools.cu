/* datatools.c - support functions for the matrix examples
 *
 * Author:  Bernd Dammann, DTU Compute
 * Version: $Revision: 1.2 $ $Date: 2015/11/10 11:03:12 $
 */
#include <stdlib.h>
#include <stdio.h>
#include <float.h>
#include <math.h>

#include "datatools.h"

void init_data(int N, double *u, double *f)
{
    int i, j;
    double x, y;
    double h = 2.0 / (N + 1);
    double *u_old;

    for (i = 0; i < N+2; i++) {
		for (j = 0; j < N+2; j++) {
			if (i == 0 || j == 0 || j == N+1) { // setting 20.0-borders
				u[i*(N+2)+j] = 20.0;
                u[i*(N+2)+j] = 20.0;
            }
            else{ // setting inner point and 0.0-border
				u[i*(N+2)+j] = 0.0;
				u_old[i*(N+2)+j] = 0.0;
			}

            x = -1 + j * h;
            y = 1 - i * h;
            // 0≤x≤1/3, −2/3≤y≤−1/3
            if (x >= 0 && x <= 1.0/3 && y >= -2.0/3 && y <= -1.0/3) {
				f[i*(N+2)+j] = 200.0;
			}
			else {
				f[i*(N+2)+j] = 0.0;
			}
        }
    }
}
