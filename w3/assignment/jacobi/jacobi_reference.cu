#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "datatools.h"
#include <stdlib.h>
#include <omp.h>
#include <hip/hip_runtime_api.h>

#define BLOCK_SIZE 16

void jacobi_cpu(int n, int num_iterations, double *f, double *u)
{

  double delta_square = 2.0 / (n + 1) * 2.0 / (n + 1);
  int k = 0, i, j;
  double *temp = NULL;
  double *u_old, *u_new;

  u_old = (double *)malloc((n + 2) * (n + 2) * sizeof(double));
  u_new = (double *)malloc((n + 2) * (n + 2) * sizeof(double));

  printf("num: %d \n k: %d \n", num_iterations, k);
#pragma omp parallel default(none) shared(n, u, u_old, num_iterations, f, delta_square, temp, k, u_new) private(j, i)
  {
#pragma omp for private(i, j)
    for (i = 0; i <= n + 1; i++)
    {
      for (j = 0; j <= n + 1; j++)
      {
        u_old[i * (n + 2) + j] = u[i * (n + 2) + j];
        u_new[i * (n + 2) + j] = u[i * (n + 2) + j];
      }
    }

    while (k < num_iterations)
    {
#pragma omp for private(i, j)
      for (i = 1; i <= n; i++)
      {
        for (j = 1; j <= n; j++)
        {
          u_new[i * (n + 2) + j] = 0.25 * (u_old[(i - 1) * (n + 2) + j] + u_old[(i + 1) * (n + 2) + j] + u_old[i * (n + 2) + (j - 1)] + u_old[i * (n + 2) + (j + 1)] + delta_square * f[i * (n + 2) + j]);
        }
      }

#pragma omp single
      {
        k += 1;
        temp = u_old;
        u_old = u_new;
        u_new = temp;
      }
    }
  }
}

__global__ void kernel1(int N, double *mat_old, double *mat_new, double *f, double delta_square)
{
  int i, j;

  for (i = 1; i < N - 1; i++)
  {
    for (j = 1; j < N - 1; j++)
    {
      mat_new[i * N + j] = (1.0 / 4.0) * (mat_old[i * N + (j - 1)] + mat_old[i * N + (j + 1)] + mat_old[(i + 1) * N + j] + mat_old[(i - 1) * N + j] + delta_square * f[i * N + j]);
    }
  }
}

void jacobi_gpu1(int N, int num_iterations, double *f, double *u)
{ // Variables declaration
  int k;
  double *h_U, *h_f, *d_U, *d_U_old, *d_U_new, *d_f, *temp_ptr;
  double ts, te;

  double delta_square = 2.0 / (N + 1) * 2.0 / (N + 1);

  // allocate memory for the necessary data fields
  hipMalloc((void **)&d_U, N * N * sizeof(double));
  hipMalloc((void **)&d_U_new, N * N * sizeof(double));
  hipMalloc((void **)&d_f, N * N * sizeof(double));
  hipHostMalloc((void **)&h_U, N * N * sizeof(double));
  hipHostMalloc((void **)&h_f, N * N * sizeof(double));

  // copy data from host to device
  hipMemcpy(d_U, h_U, N * N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_f, h_f, N * N * sizeof(double), hipMemcpyHostToDevice);

  // call kernel iterator
  // Get starting time
  ts = omp_get_wtime();

  // Assign mat_old with the initial guess (k = 0 iteration)
  d_U_old = d_U;
  for (k = 0; k < num_iterations; k++)
  {
    kernel1<<<1, 1>>>(N, d_U_old, d_U_new, d_f, delta_square);
    checkCudaErrors(hipDeviceSynchronize());

    // Swap the pointers on the CPU
    temp_ptr = d_U_old;
    d_U_old = d_U_new;
    d_U_new = temp_ptr;
  }

  // Get ending time
  te = omp_get_wtime() - ts;

  // Copy result back to host (notice that d_U_old will have the last good result!)
  hipMemcpy(h_U, d_U_old, N * N * sizeof(double), hipMemcpyDeviceToHost);

  // print results, e.g. timings, data, etc
  printf("%f\n", te);

  // Cleanup
  hipHostFree(h_U);
  hipHostFree(h_f);
  hipFree(d_f);
  hipFree(d_U);
  hipFree(d_U_new);
}

__global__ void naive_kernel(int N, double *mat_old, double *mat_new, double *f, double delta_square)
{
  int i, j;

  i = blockIdx.y * blockDim.y + threadIdx.y;
  j = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= N || j >= N)
    return;

  // Check boundary values (should be copied in the new matrix as they are)
  if (i == 0 || i == N - 1 || j == 0 || j == N - 1)
    mat_new[i * N + j] = mat_old[i * N + j];
  else
  {
    mat_new[i * N + j] = (1.0 / 4.0) * (mat_old[i * N + (j - 1)] + mat_old[i * N + (j + 1)] + mat_old[(i + 1) * N + j] + mat_old[(i - 1) * N + j] + delta_square * f[i * N + j]);
  }
}

void jacobi_gpu2(int N, int num_iterations, double *d_f, double *d_U)
{
  int k;
  double *h_U, *h_f, *d_U_old, *d_U_new, *temp_ptr;
  double ts, te;
  double delta_square = 2.0 / (N + 1) * 2.0 / (N + 1);

  // allocate memory for the necessary data fields
  hipMalloc((void **)&d_U, N * N * sizeof(double));
  hipMalloc((void **)&d_U_new, N * N * sizeof(double));
  hipMalloc((void **)&d_f, N * N * sizeof(double));
  hipHostMalloc((void **)&h_U, N * N * sizeof(double));
  hipHostMalloc((void **)&h_f, N * N * sizeof(double));

  // copy data from host to device
  hipMemcpy(d_U, h_U, N * N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_f, h_f, N * N * sizeof(double), hipMemcpyHostToDevice);

  // Define GPU thread blocks dimensions
  dim3 threads_per_block(BLOCK_SIZE, BLOCK_SIZE); // e.g. 16*16 = 256 threads in total
  dim3 num_blocks(ceil((double)N / threads_per_block.x), ceil((double)N / threads_per_block.y));

  // call kernel iterator
  // Get starting time
  ts = omp_get_wtime();

  // Assign mat_old with the initial guess (k = 0 iteration)
  d_U_old = d_U;
  for (k = 0; k < num_iterations; k++)
  {
    naive_kernel<<<num_blocks, threads_per_block>>>(N, d_U_old, d_U_new, d_f, delta_square);
    checkCudaErrors(hipDeviceSynchronize());

    // Swap the pointers on the CPU
    temp_ptr = d_U_old;
    d_U_old = d_U_new;
    d_U_new = temp_ptr;
  }
  // Get ending time
  te = omp_get_wtime() - ts;

  // Copy result back to host (notice that d_U_old will have the last good result!)
  hipMemcpy(h_U, d_U_old, N * N * sizeof(double), hipMemcpyDeviceToHost);

  // print results, e.g. timings, data, etc
  printf("%f\n", te);

  // Cleanup
  hipHostFree(h_U);
  hipHostFree(h_f);
  hipFree(d_f);
  hipFree(d_U);
  hipFree(d_U_new);
}

__global__ void kernel30(int n, double *mat_old, double *mat_new, double *f, double delta_square)
{
  int i, j;

  i = blockIdx.y * blockDim.y + threadIdx.y;
  j = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < 1 || i >= n && j < 1 && j >= n)
    return;

  if (i > 0 && i < n / 2 && j > 0 && j <= n)
  {
    mat_new[i * (n + 2) + j] = 0.25 * (mat_old[(i - 1) * (n + 2) + j] + mat_old[(i + 1) * (n + 2) + j] + mat_old[i * (n + 2) + (j - 1)] + mat_old[i * (n + 2) + (j + 1)] + delta_square * f[i * (n + 2) + j]);
  }
  else if (i == n / 2 && j > 0 && j <= n)
  {
    mat_new[i * (n + 2) + j] = 0.25 * (mat_old[(i - 1) * (n + 2) + j] + mat_old[j] + mat_old[i * (n + 2) + (j - 1)] + mat_old[i * (n + 2) + (j + 1)] + delta_square * f[i * (n + 2) + j]);
  }
}

__global__ void kernel31(int n, double *mat_old, double *mat_new, double *f, double delta_square)
{
  int i, j;

  i = blockIdx.y * blockDim.y + threadIdx.y;
  j = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < 1 || i >= n && j < 1 && j >= n)
    return;

  if (i > 0 && i < n / 2 && j > 0 && j <= n)
  {
    mat_new[i * (n + 2) + j] = 0.25 * (mat_old[(i - 1) * (n + 2) + j] + mat_old[(i + 1) * (n + 2) + j] + mat_old[i * (n + 2) + (j - 1)] + mat_old[i * (n + 2) + (j + 1)] + delta_square * f[i * (n + 2) + j]);
  }
  else if (i == n / 2 && j > 0 && j <= n)
  {
    mat_new[i * (n + 2) + j] = 0.25 * (mat_old[(i - 1) * (n + 2) + j] + mat_old[j] + mat_old[i * (n + 2) + (j - 1)] + mat_old[i * (n + 2) + (j + 1)] + delta_square * f[i * (n + 2) + j]);
  }
}

void jacobi_gpu3(int N, int num_iterations, double *h_f, double *h_U)
{
  // Variables declaration
  int k;
  double *d0_U, *d1_U, *d0_U_old, *d1_U_old, *d0_U_new, *d1_U_new, *d0_f, *d1_f, *temp_ptr;
  double delta, ts, te;

  // Allocate host memory
  hipHostMalloc((void **)&h_U, N * N * sizeof(double));
  hipHostMalloc((void **)&h_f, N * N * sizeof(double));

  double delta_square = 2.0 / (N + 1) * 2.0 / (N + 1);

  // Define GPU thread blocks dimensions
  dim3 threads_per_block(BLOCK_SIZE, BLOCK_SIZE); // e.g. 16*16 = 256 threads in total
  dim3 num_blocks(ceil((double)N / (threads_per_block.x * 2)), ceil((double)N / (threads_per_block.y * 2)));

  // Device 0
  hipSetDevice(0);
  // allocate memory for the necessary data fields
  hipMalloc((void **)&d0_U, N / 2 * N * sizeof(double));
  hipMalloc((void **)&d0_U_new, N / 2 * N * sizeof(double));
  hipMalloc((void **)&d0_f, N / 2 * N * sizeof(double));

  // copy data from host to device
  hipMemcpy(d0_U, h_U, N / 2 * N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d0_f, h_f, N / 2 * N * sizeof(double), hipMemcpyHostToDevice);

  // Device 1
  hipSetDevice(1);
  hipDeviceEnablePeerAccess(0, 0);
  // allocate memory for the necessary data fields
  hipMalloc((void **)&d1_U, N / 2 * N * sizeof(double));
  hipMalloc((void **)&d1_U_new, N / 2 * N * sizeof(double));
  hipMalloc((void **)&d1_f, N / 2 * N * sizeof(double));

  // copy data from host to device
  hipMemcpy(d1_U, h_U + N / 2, N / 2 * N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d1_f, h_f + N / 2, N / 2 * N * sizeof(double), hipMemcpyHostToDevice);

  // k1 = (n + 1) / 16 + 1;
  // k2 = (n + 1) / 32 + 1;
  // dim3 dimBlock(16, 16, 1);
  // dim3 dimGrid(k1, k2, 1);

  // Get starting time
  ts = omp_get_wtime();

  // Assign mat_old with the initial guess (k = 0 iteration)
  d0_U_old = d0_U;
  d1_U_old = d1_U;
  for (k = 0; k < num_iterations; k++)
  {
    hipSetDevice(0);
    kernel30<<<num_blocks, threads_per_block>>>(N, d0_U_old, d0_U_new, d0_f, delta);
    hipSetDevice(1);
    kernel30<<<num_blocks, threads_per_block>>>(N, d1_U_old, d1_U_new, d1_f, delta);
    checkCudaErrors(hipDeviceSynchronize());

    // Swap the pointers on the CPU
    temp_ptr = d0_U_old;
    d0_U_old = d0_U_new;
    d0_U_new = temp_ptr;

    temp_ptr = d1_U_old;
    d1_U_old = d1_U_new;
    d1_U_new = temp_ptr;
  }

  // Get ending time
  te = omp_get_wtime() - ts;

  // Copy result back to host (notice that d_U_old will have the last good result!)
  hipMemcpy(h_U, d0_U_old, N / 2 * N * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(h_U + N / 2, d1_U_old, N / 2 * N * sizeof(double), hipMemcpyDeviceToHost);

  // print results, e.g. timings, data, etc
  printf("%f\n", te);

  // Cleanup
  hipHostFree(h_U);
  hipHostFree(h_f);
  hipSetDevice(0);
  hipFree(d0_f);
  hipFree(d0_U);
  hipFree(d0_U_new);
  hipSetDevice(1);
  hipFree(d1_f);
  hipFree(d1_U);
  hipFree(d1_U_new);
}
