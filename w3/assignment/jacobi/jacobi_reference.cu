#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "datatools.h"
#include <stdlib.h>
#include <omp.h>

void jacobi_cpu(int n, int num_iterations, double *f, double *u)
{

  double delta_square = 2.0 / (n + 1) * 2.0 / (n + 1);
  int k = 0, i, j;
  double *temp = NULL;
  double *u_old, *u_new;

  u_old = (double *)malloc((n + 2) * (n + 2) * sizeof(double));
  u_new = (double *)malloc((n + 2) * (n + 2) * sizeof(double));

#pragma omp parallel default(none) shared(n, u, u_old, num_iterations, f, delta_square, temp, k, u_new) private(j, i)
  {
#pragma omp for private(i, j) schedule(dynamic)
	for (i = 0; i <= n + 1; i++)
	{
	  for (j = 0; j <= n + 1; j++)
	  {
		u_old[i * (n + 2) + j] = u[i * (n + 2) + j];
		u_new[i * (n + 2) + j] = u[i * (n + 2) + j];
	  }
	}

	while (k < num_iterations)
	{

#pragma omp for private(i, j) schedule(dynamic)
	  for (i = 1; i <= n; i++)
	  {
		for (j = 1; j <= n; j++)
		{
		  u_new[i * (n + 2) + j] = 0.25 * (u_old[(i - 1) * (n + 2) + j] + u_old[(i + 1) * (n + 2) + j] + u_old[i * (n + 2) + (j - 1)] + u_old[i * (n + 2) + (j + 1)] + delta_square * f[i * (n + 2) + j]);
		}
	  }

	  k += 1;
#pragma omp single
	  {

		temp = u_old;
		u_old = u_new;
		u_new = temp;
	  }
	}
  }
}

__global__ void kernel1(int n, double *f, double *u)
{
  int i, j;
  double delta_square = 2.0 / (n + 1) * 2.0 / (n + 1);
  double *u_old, *u_new;
  u_old = (double *)malloc((n + 2) * (n + 2) * sizeof(double));
  u_new = (double *)malloc((n + 2) * (n + 2) * sizeof(double));

  for (i = 0; i <= n + 1; i++)
  {
	for (j = 0; j <= n + 1; j++)
	{
	  u_old[i * (n + 2) + j] = u[i * (n + 2) + j];
	  u_new[i * (n + 2) + j] = u[i * (n + 2) + j];
	}
  }

  for (i = 1; i <= n; i++)
  {
	for (j = 1; j <= n; j++)
	{
	  u_new[i * (n + 2) + j] = 0.25 * (u_old[(i - 1) * (n + 2) + j] + u_old[(i + 1) * (n + 2) + j] + u_old[i * (n + 2) + (j - 1)] + u_old[i * (n + 2) + (j + 1)] + delta_square * f[i * (n + 2) + j]);
	}
  }
}

void jacobi_gpu1(int n, int num_iterations, double *f, double *u)
{
  int k = 0;
  double *u_gpu, *u_old_gpu, *u_new_gpu, *f_gpu;
  double *u_old, *u_new;
  u_old = (double *)malloc((n + 2) * (n + 2) * sizeof(double));
  u_new = (double *)malloc((n + 2) * (n + 2) * sizeof(double));
  double *temp = NULL;

  hipMalloc((void **)&u_gpu, (n + 2) * (n + 2) * sizeof(double));
  hipMalloc((void **)&u_old_gpu, (n + 2) * (n + 2) * sizeof(double));
  hipMalloc((void **)&u_new_gpu, (n + 2) * (n + 2) * sizeof(double));
  hipMalloc((void **)&f_gpu, (n + 2) * (n + 2) * sizeof(double));
  //get time
  hipMemcpy(u_gpu, u, (n + 2) * (n + 2) * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(u_old_gpu, u_old, (n + 2) * (n + 2) * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(u_new_gpu, u_new, (n + 2) * (n + 2) * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(f_gpu, f, (n + 2) * (n + 2) * sizeof(double), hipMemcpyHostToDevice);
  //get time
  while (k < num_iterations)
  {
	kernel1 <<<1, 1>>> (n, f_gpu, u_gpu);
	hipDeviceSynchronize();
	temp = u_old_gpu;
	u_old_gpu = u_new_gpu;
	u_new_gpu = temp;
	k += 1;
	//get time
  }
  hipMemcpy(u, u_gpu, (n + 2) * (n + 2) * sizeof(double), hipMemcpyDeviceToHost);
  //get time
  hipFree(u_gpu);
  hipFree(u_old_gpu);
  hipFree(u_new_gpu);
  hipFree(f_gpu);
}

__global__ void naive_kernel(int n, double *f, double *u)
{
  int i, j;
  double delta_square = 2.0 / (n + 1) * 2.0 / (n + 1);
  double *u_old, *u_new;
  u_old = (double *)malloc((n + 2) * (n + 2) * sizeof(double));
  u_new = (double *)malloc((n + 2) * (n + 2) * sizeof(double));

  i = blockIdx.y * blockDim.y + threadIdx.y;
  j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i > 0 && i <= n && j > 0 && j <= n)
  {
	u_old[i * (n + 2) + j] = u[i * (n + 2) + j];
	u_new[i * (n + 2) + j] = u[i * (n + 2) + j];
  }
  if (i > 0 && i <= n && j > 0 && j <= n)
  {
	u_new[i * (n + 2) + j] = 0.25 * (u_old[(i - 1) * (n + 2) + j] + u_old[(i + 1) * (n + 2) + j] + u_old[i * (n + 2) + (j - 1)] + u_old[i * (n + 2) + (j + 1)] + delta_square * f[i * (n + 2) + j]);
  }
}

void jacobi_gpu2(int n, int num_iterations, double *f, double *u)
{
  int k = 0;
  int k1, k2;
  double *u_gpu, *u_old_gpu, *u_new_gpu, *f_gpu;
  double *u_old, *u_new;
  u_old = (double *)malloc((n + 2) * (n + 2) * sizeof(double));
  u_new = (double *)malloc((n + 2) * (n + 2) * sizeof(double));
  double *temp = NULL;

  hipMalloc((void **)&u_gpu, (n + 2) * (n + 2) * sizeof(double));
  hipMalloc((void **)&u_old_gpu, (n + 2) * (n + 2) * sizeof(double));
  hipMalloc((void **)&u_new_gpu, (n + 2) * (n + 2) * sizeof(double));
  hipMalloc((void **)&f_gpu, (n + 2) * (n + 2) * sizeof(double));

  hipMemcpy(u_gpu, u, (n + 2) * (n + 2) * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(u_old_gpu, u_old, (n + 2) * (n + 2) * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(u_new_gpu, u_new, (n + 2) * (n + 2) * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(f_gpu, f, (n + 2) * (n + 2) * sizeof(double), hipMemcpyHostToDevice);

  k1 = (n + 1) / 16 + 1;
  k2 = (n + 1) / 16 + 1;
  dim3 dimBlock(16, 16, 1);
  dim3 dimGrid(k1, k2, 1);

  while (k < num_iterations)
  {
	naive_kernel<<<dimGrid, dimBlock>>>(n, f_gpu, u_gpu);
	hipDeviceSynchronize();
	temp = u_old_gpu;
	u_old_gpu = u_new_gpu;
	u_new_gpu = temp;
	k += 1;
  }
  hipMemcpy(u, u_gpu, (n + 2) * (n + 2) * sizeof(double), hipMemcpyDeviceToHost);
  //get time
  hipFree(u_gpu);
  hipFree(u_old_gpu);
  hipFree(u_new_gpu);
  hipFree(f_gpu);
}

__global__ void kernel30(int n, double *f, double *u, double *u_old_v2)
{
  int i, j;
  double delta_square = 2.0 / (n + 1) * 2.0 / (n + 1);
  double *u_old, *u_new;
  u_old = (double *)malloc((n + 2) * (n + 2) * sizeof(double));
  u_new = (double *)malloc((n + 2) * (n + 2) * sizeof(double));

  i = blockIdx.y * blockDim.y + threadIdx.y;
  j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i > 0 && i <= n && j > 0 && j <= n)
  {
	u_old[i * (n + 2) + j] = u[i * (n + 2) + j];
	u_new[i * (n + 2) + j] = u[i * (n + 2) + j];
  }
  if (i > 0 && i < n / 2 && j > 0 && j <= n)
  {
	u_new[i * (n + 2) + j] = 0.25 * (u_old[(i - 1) * (n + 2) + j] + u_old[(i + 1) * (n + 2) + j] + u_old[i * (n + 2) + (j - 1)] + u_old[i * (n + 2) + (j + 1)] + delta_square * f[i * (n + 2) + j]);
  }
  else if (i == n / 2 && j > 0 && j <= n)
  {
	u_new[i * (n + 2) + j] = 0.25 * (u_old[(i - 1) * (n + 2) + j] + u_old_v2[j] + u_old[i * (n + 2) + (j - 1)] + u_old[i * (n + 2) + (j + 1)] + delta_square * f[i * (n + 2) + j]);
  }
}

__global__ void kernel31(int n, double *f, double *u, double *u_old_v2)
{
  int i, j;
  double delta_square = 2.0 / (n + 1) * 2.0 / (n + 1);
  double *u_old, *u_new;
  u_old = (double *)malloc((n + 2) * (n + 2) * sizeof(double));
  u_new = (double *)malloc((n + 2) * (n + 2) * sizeof(double));

  i = blockIdx.y * blockDim.y + threadIdx.y;
  j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i > 0 && i <= n && j > 0 && j <= n)
  {
	u_old[i * (n + 2) + j] = u[i * (n + 2) + j];
	u_new[i * (n + 2) + j] = u[i * (n + 2) + j];
  }
  if (i > 0 && i < n / 2 && j > 0 && j <= n)
  {
	u_new[i * (n + 2) + j] = 0.25 * (u_old[(i - 1) * (n + 2) + j] + u_old[(i + 1) * (n + 2) + j] + u_old[i * (n + 2) + (j - 1)] + u_old[i * (n + 2) + (j + 1)] + delta_square * f[i * (n + 2) + j]);
  }
  else if (i == 0 && j > 0 && j <= n)
  {
	u_new[i * (n + 2) + j] = 0.25 * (u_old_v2[n / 2 * (n + 2) + j] + u_old[(i + 1) * (n + 2) + j] + u_old[i * (n + 2) + (j - 1)] + u_old[i * (n + 2) + (j + 1)] + delta_square * f[i * (n + 2) + j]);
  }
}

void jacobi_gpu3(int n, int num_iterations, double *f, double *u)
{
  int k = 0;
  int k1, k2;
  double *u_gpu0, *u_old_gpu0, *u_new_gpu0, *f_gpu0, *u_gpu1, *u_old_gpu1, *u_new_gpu1, *f_gpu1;
  double *u_old, *u_new;
  u_old = (double *)malloc((n + 2) * (n + 2) * sizeof(double));
  u_new = (double *)malloc((n + 2) * (n + 2) * sizeof(double));
  double *temp0 = NULL;
  double *temp1 = NULL;
  hipSetDevice(0);
  hipMalloc((void **)&u_gpu0, (n + 2) * (n + 2) * sizeof(double) / 2);
  hipMalloc((void **)&u_old_gpu0, (n + 2) * (n + 2) * sizeof(double) / 2);
  hipMalloc((void **)&f_gpu0, (n + 2) * (n + 2) * sizeof(double) / 2);
  hipMalloc((void **)&u_new_gpu0, (n + 2) * (n + 2) * sizeof(double) / 2);

  hipSetDevice(1);
  hipMalloc((void **)&u_gpu1, (n + 2) * (n + 2) * sizeof(double) / 2);
  hipMalloc((void **)&u_old_gpu1, (n + 2) * (n + 2) * sizeof(double) / 2);
  hipMalloc((void **)&f_gpu1, (n + 2) * (n + 2) * sizeof(double) / 2);
  hipMalloc((void **)&u_new_gpu1, (n + 2) * (n + 2) * sizeof(double) / 2);

  hipSetDevice(0);
  hipMemcpy(u_gpu0, u, (n + 2) * (n + 2) * sizeof(double) / 2, hipMemcpyHostToDevice);
  hipMemcpy(u_old_gpu0, u_old, (n + 2) * (n + 2) * sizeof(double) / 2, hipMemcpyHostToDevice);
  hipMemcpy(u_new_gpu0, u_new, (n + 2) * (n + 2) * sizeof(double) / 2, hipMemcpyHostToDevice);
  hipMemcpy(f_gpu0, f, (n + 2) * (n + 2) * sizeof(double) / 2, hipMemcpyHostToDevice);
  hipDeviceEnablePeerAccess(1, 0);

  hipSetDevice(1);
  hipMemcpy(u_gpu1, u + (n + 2) * (n + 2) / 2, (n + 2) * (n + 2) * sizeof(double) / 2, hipMemcpyHostToDevice);
  hipMemcpy(u_old_gpu1, u_old + (n + 2) * (n + 2) / 2, (n + 2) * (n + 2) * sizeof(double) / 2, hipMemcpyHostToDevice);
  hipMemcpy(f_gpu1, f + (n + 2) * (n + 2) / 2, (n + 2) * (n + 2) * sizeof(double) / 2, hipMemcpyHostToDevice);
  hipMemcpy(u_new_gpu1, u_new + (n + 2) * (n + 2) / 2, (n + 2) * (n + 2) * sizeof(double) / 2, hipMemcpyHostToDevice);
  hipDeviceEnablePeerAccess(0, 0);

  k1 = (n + 1) / 16 + 1;
  k2 = (n + 1) / 32 + 1;
  dim3 dimBlock(16, 16, 1);
  dim3 dimGrid(k1, k2, 1);

  while (k < num_iterations)
  {
	hipSetDevice(0);
	kernel30<<<dimGrid, dimBlock>>>(n, f_gpu0, u_gpu0, u_old_gpu1);

	hipSetDevice(1);
	kernel31<<<dimGrid, dimBlock>>>(n, f_gpu1, u_gpu1, u_old_gpu0);
	hipDeviceSynchronize();

	hipSetDevice(0);
	hipDeviceSynchronize();

	temp0 = u_old_gpu0;
	u_old_gpu0 = u_new_gpu0;
	u_new_gpu0 = temp0;

	temp1 = u_old_gpu1;
	u_old_gpu1 = u_new_gpu1;
	u_new_gpu1 = temp1;
	k += 1;
  }
  hipSetDevice(0);
  hipMemcpy(u, u_gpu0, (n + 2) * (n + 2) * sizeof(double) / 2, hipMemcpyDeviceToHost);
  hipFree(u_gpu0);
  hipFree(u_old_gpu0);
  hipFree(f_gpu0);
  hipFree(u_new_gpu0);

  hipSetDevice(1);
  hipMemcpy(u + (n + 2) * (n + 2) / 2, u_gpu1, (n + 2) * (n + 2) * sizeof(double) / 2, hipMemcpyDeviceToHost);
  //t6 = omp_get_wtime();
  hipFree(u_gpu1);
  hipFree(u_old_gpu1);
  hipFree(f_gpu1);
  hipFree(u_new_gpu1);
}
