extern "C" {

#include <hip/hip_runtime.h>
#include <stdio.h>
}

#define nb_elt 10

__global__
void gpu1(int m, int n, int k, double *a, double *b, double *c){
    int i, j, p;

    for(i = 0; i < m*n; i++){
        c[i] = 0.0;
    }

    for(i = 0; i < m; i++){
        for(p = 0; p < k; p++){
            for(j = 0; j < m; j++){
                c[i*n+j] += a[i*k+p]*b[p*n+j];
            }
        }
    }
}


__global__
void gpu2(int m, int n, int k, double *a, double *b, double *c){
    int x, y, i;
    double sum = 0.0;

    x = blockIdx.x*blockDim.x + threadIdx.x; // col
    y = blockIdx.y*blockDim.y + threadIdx.y; // row

    for(i = 0; i < k; i++){
        sum += a[y*k+i]*b[i*n + x];
    }

    c[y*k + x] = sum;
}

__global__
void gpu3(int m, int n, int k, double *a, double *b, double *c){
    int x, y, i;
    double sum1 = 0.0, sum2 = 0.0;

    x = blockIdx.x*blockDim.x + threadIdx.x; // col
    y = blockIdx.y*blockDim.y + threadIdx.y; // row

    if( y == n-1){
        // if we are in the last row we only compute one element
        for(i = 0; i < k; i++){
            sum1 += a[y*k+i]*b[i*n + x];
        }
        c[y*k + x] = sum1;

    }else{
        // otherwise we can compute two elements
        for(i = 0; i < k; i++){
            sum1 += a[y*k+i]*b[i*n + x];
            sum2 += a[y*k+i+1]*b[i*n + x];
        }

        c[y*k + x] = sum1;
        c[y*k + x+1] = sum2;
    }
}


__global__
void gpu4(int m, int n, int k, double *a, double *b, double *c){
    int i,j,l,t;

    i = nb_elt*(blockIdx.y*blockDim.y+threadIdx.y);
    j = blockIdx.x*blockDim.x+threadIdx.x;

    if (i < m-nb_elt && j < n) {
        for (t = 0; t < nb_elt; t++) {
            for (l = 0; l < k; l++) {
                c[(i+t)*n+j] += a[(i+t)*k+l]*b[l*n+j];
            }
        }
    }
    else if (i < m && j < n) {
        for (t = 0; t < nb_elt; t++) {
            if (i+t < m) {
                for (l = 0; l < k; l++) {
                    c[(i+t)*n+j] += a[(i+t)*k+l]*b[l*n+j];
                }
            }
        }
    }
}


__global__
void kernel5(int m, int n, int k, double *a, double *b, double *c) {

	int i,j,r,s;

	double c_val = 0.0;

	i = blockIdx.y*blockDim.y + threadIdx.y; // row in C
	j = blockIdx.x*blockDim.x + threadIdx.x; // col in C

	for (r = 0; r < k/blockDim.x; r++) {

		__shared__ double a_blk[16][16];
		__shared__ double b_blk[16][16];

		a_blk[threadIdx.y][threadIdx.x] = a[i*k+r*blockDim.x+threadIdx.x];
		b_blk[threadIdx.y][threadIdx.x] = b[(r*blockDim.y+threadIdx.y)*n+j];

		__syncthreads(); // submatrices loaded before starting computation

		for (s = 0; s < blockDim.x; s++) {
			c_val += a_blk[threadIdx.y][s]*b_blk[s][threadIdx.x];
		}

		__syncthreads();
	}
	c[i*n+j] = c_val;
}
