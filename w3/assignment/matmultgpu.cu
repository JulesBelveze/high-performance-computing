
#include <hip/hip_runtime.h>
#define NB_ELT 11

__global__
void gpu1(int m, int n, int k, double *a, double *b, double *c){
    int i, j, p;

    for (i = 0; i < m*n; i++){
        c[i] = 0.0;
    }

    for(i = 0; i < m; i++){
        for(p = 0; p < k; p++){
            for(j = 0; j < n; j++){
                c[i*n+j] += a[i*k+p]*b[p*n+j];
            }
        }
    }
}


__global__
void gpu2(int m, int n, int k, double *a, double *b, double *c){
    int x, y, i;
    double sum = 0.0;

    x = blockIdx.x*blockDim.x + threadIdx.x; // col
    y = blockIdx.y*blockDim.y + threadIdx.y; // row

    if (y < m && x < n) {
        for(i = 0; i < k; i++){
            sum += a[y*k+i]*b[i*n + x];
        }
        c[y*n + x] = sum;
    }
}

__global__
void gpu3(int m, int n, int k, double *a, double *b, double *c){
    int x, y, i;
	double sum=0.0, sum2=0.0;

	y = blockIdx.y*blockDim.y + threadIdx.y;
	x = 2*(blockIdx.x*blockDim.x + threadIdx.x);

	if ((y < m) && (x < n-1)) {
		for(i = 0; i < k; i++){
			sum += a[y*k+i]*b[i*n + x];
			sum2 += a[y*k+i]*b[i*n + x +1];
			}
		c[y*n+x] = sum;
		c[y*n+x+1] = sum2;
	}
	else if ((y < m) && (x == n-1)){
		for(i = 0; i < k; i++){
			sum += a[y*k+i]*b[i*n + x];
			}
		c[y*n+x] = sum;
	}
}


__global__
void gpu4(int m, int n, int k, double *a, double *b, double *c){
    int x,y,l,t;


    y = NB_ELT*(blockIdx.y*blockDim.y+threadIdx.y);
    x = blockIdx.x*blockDim.x+threadIdx.x;

    if (y < m-NB_ELT && x < n) {
        for (t = 0; t < NB_ELT; t++) {
            for (l = 0; l < k; l++) {
                c[(y+t)*n+x] += a[(y+t)*k+l]*b[l*n+x];
            }
        }
    }
    else if (y < m && x < n) {
        for (t = 0; t < NB_ELT; t++) {
            if (y+t < m) {
                for (l = 0; l < k; l++) {
                    c[(y+t)*n+x] += a[(y+t)*k+l]*b[l*n+x];
                }
            }
        }
    }
}


__global__
void gpu5(int m, int n, int k, double *a, double *b, double *c) {

	int i,j,r,s;

	double c_val = 0.0;

	i = blockIdx.y*blockDim.y + threadIdx.y; // row in C
	j = blockIdx.x*blockDim.x + threadIdx.x; // col in C

	for (r = 0; r < k/blockDim.x; r++) {

		__shared__ double a_blk[16][16];
		__shared__ double b_blk[16][16];

		a_blk[threadIdx.y][threadIdx.x] = a[i*k+r*blockDim.x+threadIdx.x];
		b_blk[threadIdx.y][threadIdx.x] = b[(r*blockDim.y+threadIdx.y)*n+j];

		__syncthreads(); // submatrices loaded before starting computation

		for (s = 0; s < blockDim.x; s++) {
			c_val += a_blk[threadIdx.y][s]*b_blk[s][threadIdx.x];
		}

		__syncthreads();
	}
	c[i*n+j] = c_val;
}
