#include "hip/hip_runtime.h"
// CPU version 3

__global__
void gpu3(int m, int n, int k, double *a, double *b, double *c){
    int x, y;
    double sum1 = 0.0, sum2 = 0.0;

    x = blockIdx.x*blockDim.x + threadIdx.x; // col
    y = blockIdx.y*blockDim.y + threadIdx.y; // row

    if( y == n-1){
        // if we are in the last row we only compute one element
        for(i = 0; i < k; i++){
            sum1 += a[y*k+i]*b[i*n + x];
        }
        c[y*k + x] = sum1;

    }else{
        // otherwise we can compute two elements
        for(i = 0; i < k; i++){
            sum1 += a[y*k+i]*b[i*n + x];
            sum2 += a[y*k+i+1]*b[i*n + x];
        }

        c[y*k + x] = sum1;
        c[y*k + x+1] = sum2;
    }
}

void matmult_gpu3(int m, int n, int k, double *a, double *b, double *c){
    double *d_a, *d_b, *d_c;
    int k1, k2;

    // preparing GPU grid and block
    k1 = (n-1)/16+1;
    k2 = (m-1)/16+1;
    dim3 dimBlock(16,16,1);
	dim3 dimGrid((k1-1)/2+1,k2,1);

    // allocate space for device copies
    hipMalloc((void **)&d_a, n*k*sizeof(double));
    hipMalloc((void **)&d_b, k*m*sizeof(double));
    hipMalloc((void **)&d_c, m*n*sizeof(double));

    // copying input to device
    hipMemcpy(d_a, a, n*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k*m*sizeof(double), hipMemcpyHostToDevice);

    gpu2<<<dimGrid,dimBlock>>>(m, n, k, d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}
