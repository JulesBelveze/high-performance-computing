
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <hipblas.h>

extern "C" {
#include <cblas.h>

// CBLAS version
void matmult_lib(int m, int n, int k, double *a, double *b, double *c) {
    cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, 1.0, \
                                        a, k, b, n, 0.0, c, n);
}

void
matmult_gpulib(int m, int n, int k, double *a, double *b, double *c) {

    const double alf = 1.0;
    const double bet = 0.0;

    const double *alpha = &alf;
    const double *beta = &bet;

    hipblasHandle_t handle;
    //cublasOperation_t transa = "n";
    //cublasOperation_t transb = "n";
    hipblasCreate(&handle);
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, b, n, a, k, beta, c, n);
    hipblasDestroy(handle);
}

}
