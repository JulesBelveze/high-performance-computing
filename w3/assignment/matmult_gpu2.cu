#include "hip/hip_runtime.h"
// GPU Sequential version v2

__global__
void gpu2(int m, int n, int k, double *a, double *b, double *c){
    int x, y;
    double sum = 0.0;

    x = blockIdx.x*blockDim.x + threadIdx.x; // col
    y = blockIdx.y*blockDim.y + threadIdx.y; // row

    for(i = 0; i < k; i++){
        sum += a[y*k+i]*b[i*n + x];
    }

    c[y*k + x] = sum;
}


void matmult_gpu2(int m, int n, int k, double *a, double *b, double *c){
    double *d_a, *d_b, *d_c;
    int k1, k2;

    // preparing GPU grid and block
    k1 = (n-1)/16+1;
    k2 = (m-1)/16+1;
    dim3 dimBlock(16,16,1);
    dim3 dimGrid(k1,k2,1);

    // allocate space for device copies
    hipMalloc((void **)&d_a, n*k*sizeof(double));
    hipMalloc((void **)&d_b, k*m*sizeof(double));
    hipMalloc((void **)&d_c, m*n*sizeof(double));

    // copying input to device
    hipMemcpy(d_a, a, n*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k*m*sizeof(double), hipMemcpyHostToDevice);

    gpu2<<<dimGrid,dimBlock>>>(m, n, k, d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}
