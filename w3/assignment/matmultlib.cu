extern "C" {
#include <cblas.h>
#include <math.h>
#include <stdio.h>
#include <omp.h>
}
#include "matmultgpu.h"
#include <hip/hip_runtime_api.h>
#include "hipblas.h"
#define nb_elt 8

extern "C" {

// cblas dgemm
void matmult_lib(int m, int n, int k, double *A, double *B, double *C)
{
    cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, 1, A, k, B, n, 0, C, n);
}

void matmult_gpu1(int m, int n, int k,  double *a, double *b, double *c){
    double *d_a, *d_b, *d_c;

    // allocate space for device copies
    hipMalloc((void **)&d_a, m*k*sizeof(double));
    hipMalloc((void **)&d_b, k*n*sizeof(double));
    hipMalloc((void **)&d_c, m*n*sizeof(double));

    // copying input to device
    hipMemcpy(d_a, a, n*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k*m*sizeof(double), hipMemcpyHostToDevice);

    double time = omp_get_wtime();
    gpu1<<<1,1>>>(m, n, k, d_a, d_b, d_c);

    hipDeviceSynchronize();
    double elapsed = omp_get_wtime() - time;

    printf("%f \n", elapsed);

    // copying output to host
    hipMemcpy(c, d_c, m*n*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}


void matmult_gpu2(int m, int n, int k, double *a, double *b, double *c){
    double *d_a, *d_b, *d_c;
    int k1, k2;

    // preparing GPU grid and block
    k1 = (n-1)/16+1;
    k2 = (m-1)/16+1;
    dim3 dimBlock(16,16,1);
    dim3 dimGrid(k1,k2,1);

    // allocate space for device copies
    hipMalloc((void **)&d_a, m*k*sizeof(double));
    hipMalloc((void **)&d_b, k*n*sizeof(double));
    hipMalloc((void **)&d_c, m*n*sizeof(double));

    // copying input to device
    hipMemcpy(d_a, a, m*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k*n*sizeof(double), hipMemcpyHostToDevice);

    gpu2<<<dimGrid,dimBlock>>>(m, n, k, d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}


void matmult_gpu3(int m, int n, int k, double *a, double *b, double *c){
    double *d_a, *d_b, *d_c;
    int k1, k2;

    // preparing GPU grid and block
    k1 = (n-1)/16+1;
    k2 = (m-1)/16+1;
    dim3 dimBlock(16,16,1);
	dim3 dimGrid((k1-1)/2+1,k2,1);

    // allocate space for device copies
    hipMalloc((void **)&d_a, m*k*sizeof(double));
    hipMalloc((void **)&d_b, k*n*sizeof(double));
    hipMalloc((void **)&d_c, m*n*sizeof(double));

    // copying input to device
    hipMemcpy(d_a, a, m*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k*n*sizeof(double), hipMemcpyHostToDevice);

    gpu3<<<dimGrid,dimBlock>>>(m, n, k, d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}


void matmult_gpu4(int m, int n, int k, double *a, double *b, double *c){
    double *d_a, *d_b, *d_c;
    int k1, k2;

    // preparing GPU grid and block
    k1 = (n-1)/16+1;
    k2 = (m-1)/16+1;
    dim3 dimBlock(16,16,1);
	dim3 dimGrid(k1,(k2-1)/nb_elt+1,1);

    // allocate space for device copies
    hipMalloc((void **)&d_a, m*k*sizeof(double));
    hipMalloc((void **)&d_b, k*n*sizeof(double));
    hipMalloc((void **)&d_c, m*n*sizeof(double));

    // copying input to device
    hipMemcpy(d_a, a, m*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k*n*sizeof(double), hipMemcpyHostToDevice);

    gpu4<<<dimGrid,dimBlock>>>(m, n, k, d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}


void
matmult_gpu5(int m, int n, int k, double *a, double *b, double *c) {

    double *d_a, *d_b, *d_c;
    int k1, k2;

    hipMalloc((void **) &d_a, m * k * sizeof(double));
    hipMalloc((void **) &d_b, k * n * sizeof(double));
    hipMalloc((void **) &d_c, m * n * sizeof(double));

    hipMemcpy(d_a, a, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k * n * sizeof(double), hipMemcpyHostToDevice);

    k1 = (n-1)/16+1;
    k2 = (m-1)/16+1;
    dim3 dimBlock(16,16,1);
    dim3 dimGrid(k1,k2,1);

    gpu5<<<dimGrid,dimBlock>>>(m, n, k, d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}

void
matmult_gpulib(int m, int n, int k, double *a, double *b, double *c) {
    const double alpha = 1.0, beta = 0.0;
    double *d_A, *d_B, *d_C;

    // Allocate memory on device
    hipMalloc((void **)&d_A, m * k * sizeof(double));
    hipMalloc((void **)&d_B, k * n * sizeof(double));
    hipMalloc((void **)&d_C, m * n * sizeof(double));

    if (d_A == NULL || d_B == NULL || d_C == NULL)
    {
        fprintf(stderr, "memory allocation failed!\n");
        return;
    }

    // Copy data from host to device
    hipMemcpy(d_A, a, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, k * n * sizeof(double), hipMemcpyHostToDevice);


    // Create handle for CUBLAS
    hipblasHandle_t handle;
    if(hipblasCreate(&handle)!=HIPBLAS_STATUS_SUCCESS)
    {
        printf("Error initializing CUDA runtime.\n");
        return;
    }

    // Kernel invocation
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B, n, d_A, k, &beta, d_C, n);

    // Destroy handle
    hipblasDestroy(handle);

    checkCudaErrors(hipDeviceSynchronize());

    // Copy result back to host
    hipMemcpy(c, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
}
