extern "C" {
#include <cblas.h>
#include <math.h>
#include <stdio.h>
#include <omp.h>
}
#include "matmultgpu.h"
#include <hip/hip_runtime_api.h>
#include "hipblas.h"

extern "C" {
#define NB_ELT 11

// cblas dgemm
void matmult_lib(int m, int n, int k, double *A, double *B, double *C)
{
    cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, 1, A, k, B, n, 0, C, n);
}

void matmult_gpu1(int m, int n, int k,  double *a, double *b, double *c){
    double *d_a, *d_b, *d_c;

    // allocate space for device copies
    hipMalloc((void **)&d_a, m*k*sizeof(double));
    hipMalloc((void **)&d_b, k*n*sizeof(double));
    hipMalloc((void **)&d_c, m*n*sizeof(double));

    // copying input to device
    hipMemcpy(d_a, a, n*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k*m*sizeof(double), hipMemcpyHostToDevice);

    double time = omp_get_wtime();
    gpu1<<<1,1>>>(m, n, k, d_a, d_b, d_c);

    hipDeviceSynchronize();
    double elapsed = omp_get_wtime() - time;

    printf("%f \n", elapsed);

    // copying output to host
    hipMemcpy(c, d_c, m*n*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}


void matmult_gpu2(int m, int n, int k, double *a, double *b, double *c){
    double *d_a, *d_b, *d_c;
    int k1, k2;

    // preparing GPU grid and block
    k1 = (n-1)/16+1;
    k2 = (m-1)/16+1;
    dim3 dimBlock(16,16,1);
    dim3 dimGrid(k1,k2,1);

    // allocate space for device copies
    hipMalloc((void **)&d_a, m*k*sizeof(double));
    hipMalloc((void **)&d_b, k*n*sizeof(double));
    hipMalloc((void **)&d_c, m*n*sizeof(double));

    // copying input to device
    hipMemcpy(d_a, a, m*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k*n*sizeof(double), hipMemcpyHostToDevice);

    gpu2<<<dimGrid,dimBlock>>>(m, n, k, d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}


void matmult_gpu3(int m, int n, int k, double *a, double *b, double *c){
    double *d_a, *d_b, *d_c;
    int k1, k2;

    // preparing GPU grid and block
    k1 = (n-1)/16+1;
    k2 = (m-1)/16+1;
    dim3 dimBlock(16,16,1);
	dim3 dimGrid((k1-1)/2+1,k2,1);

    // allocate space for device copies
    hipMalloc((void **)&d_a, m*k*sizeof(double));
    hipMalloc((void **)&d_b, k*n*sizeof(double));
    hipMalloc((void **)&d_c, m*n*sizeof(double));

    // copying input to device
    hipMemcpy(d_a, a, m*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k*n*sizeof(double), hipMemcpyHostToDevice);

    gpu3<<<dimGrid,dimBlock>>>(m, n, k, d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}


void matmult_gpu4(int m, int n, int k, double *a, double *b, double *c){
    double *d_a, *d_b, *d_c;
    int k1, k2;

    // preparing GPU grid and block
    k1 = (n-1)/16+1;
    k2 = (m-1)/16+1;
    dim3 dimBlock(16,16,1);
	dim3 dimGrid(k1,(k2-1)/NB_ELT+1,1);

    // allocate space for device copies
    hipMalloc((void **)&d_a, m*k*sizeof(double));
    hipMalloc((void **)&d_b, k*n*sizeof(double));
    hipMalloc((void **)&d_c, m*n*sizeof(double));

    // copying input to device
    hipMemcpy(d_a, a, m*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k*n*sizeof(double), hipMemcpyHostToDevice);

    gpu4<<<dimGrid,dimBlock>>>(m, n, k, d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}


void
matmult_gpu5(int m, int n, int k, double *a, double *b, double *c) {

    double *d_a, *d_b, *d_c;
    int k1, k2;

    hipMalloc((void **) &d_a, m * k * sizeof(double));
    hipMalloc((void **) &d_b, k * n * sizeof(double));
    hipMalloc((void **) &d_c, m * n * sizeof(double));

    hipMemcpy(d_a, a, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k * n * sizeof(double), hipMemcpyHostToDevice);

    k1 = (n-1)/16+1;
    k2 = (m-1)/16+1;
    dim3 dimBlock(16,16,1);
    dim3 dimGrid(k1,k2,1);

    gpu5<<<dimGrid,dimBlock>>>(m, n, k, d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}

void
matmult_gpulib(int m, int n, int k, double *a, double *b, double *c) {
    const double alpha = 1.0, beta = 0.0;
    double *d_a, *d_b, *d_c;

    hipMalloc((void **)&d_a, m * k * sizeof(double));
    hipMalloc((void **)&d_b, k * n * sizeof(double));
    hipMalloc((void **)&d_c, m * n * sizeof(double));

    if (d_a == NULL || d_b == NULL || d_c == NULL)
    {
        fprintf(stderr, "memory allocation failed!\n");
        return;
    }

    hipMemcpy(d_a, a, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k * n * sizeof(double), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    if(hipblasCreate(&handle)!=HIPBLAS_STATUS_SUCCESS)
    {
        printf("Error initializing CUDA runtime.\n");
        return;
    }

    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_b, n, d_a, k, &beta, d_c, n);

    hipblasDestroy(handle);

    checkCudaErrors(hipDeviceSynchronize());

    hipMemcpy(c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}
}
