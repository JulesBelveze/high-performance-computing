extern "C" {
#include <cblas.h>
#include <math.h>
#include <stdio.h>
#include <omp.h>
}
#include "matmultgpu.h"
#include <hip/hip_runtime_api.h>
#include "hipblas.h"
#define nb_elt 10

extern "C" {

// cblas dgemm
void matmult_lib(int m, int n, int k, double *A, double *B, double *C)
{
    cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, 1, A, k, B, n, 0, C, n);
}

void matmult_gpu1(int m, int n, int k,  double *a, double *b, double *c){
    double *d_a, *d_b, *d_c;

    // allocate space for device copies
    hipMalloc((void **)&d_a, n*k*sizeof(double));
    hipMalloc((void **)&d_b, k*m*sizeof(double));
    hipMalloc((void **)&d_c, m*n*sizeof(double));

    // copying input to device
    hipMemcpy(d_a, a, n*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k*m*sizeof(double), hipMemcpyHostToDevice);

    double time = omp_get_wtime();
    gpu1<<<1,1>>>(m, n, k, d_a, d_b, d_c);

    hipDeviceSynchronize();
    double elapsed = omp_get_wtime() - time;

    // copying output to host
    hipMemcpy(c, d_c, n*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}


void matmult_gpu2(int m, int n, int k, double *a, double *b, double *c){
    double *d_a, *d_b, *d_c;
    int k1, k2;

    // preparing GPU grid and block
    k1 = (n-1)/16+1;
    k2 = (m-1)/16+1;
    dim3 dimBlock(16,16,1);
    dim3 dimGrid(k1,k2,1);

    // allocate space for device copies
    hipMalloc((void **)&d_a, n*k*sizeof(double));
    hipMalloc((void **)&d_b, k*m*sizeof(double));
    hipMalloc((void **)&d_c, m*n*sizeof(double));

    // copying input to device
    hipMemcpy(d_a, a, n*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k*m*sizeof(double), hipMemcpyHostToDevice);

    gpu2<<<dimGrid,dimBlock>>>(m, n, k, d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}


void matmult_gpu3(int m, int n, int k, double *a, double *b, double *c){
    double *d_a, *d_b, *d_c;
    int k1, k2;

    // preparing GPU grid and block
    k1 = (n-1)/16+1;
    k2 = (m-1)/16+1;
    dim3 dimBlock(16,16,1);
	dim3 dimGrid((k1-1)/2+1,k2,1);

    // allocate space for device copies
    hipMalloc((void **)&d_a, n*k*sizeof(double));
    hipMalloc((void **)&d_b, k*m*sizeof(double));
    hipMalloc((void **)&d_c, m*n*sizeof(double));

    // copying input to device
    hipMemcpy(d_a, a, n*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k*m*sizeof(double), hipMemcpyHostToDevice);

    gpu3<<<dimGrid,dimBlock>>>(m, n, k, d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}


void matmult_gpu4(int m, int n, int k, double *a, double *b, double *c){
    double *d_a, *d_b, *d_c;
    int k1, k2;

    // preparing GPU grid and block
    k1 = (n-1)/16+1;
    k2 = (m-1)/16+1;
    dim3 dimBlock(16,16,1);
	dim3 dimGrid(k1,(k2-1)/nb_elt+1,1);

    // allocate space for device copies
    hipMalloc((void **)&d_a, n*k*sizeof(double));
    hipMalloc((void **)&d_b, k*m*sizeof(double));
    hipMalloc((void **)&d_c, m*n*sizeof(double));

    // copying input to device
    hipMemcpy(d_a, a, n*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k*m*sizeof(double), hipMemcpyHostToDevice);

    gpu4<<<dimGrid,dimBlock>>>(m, n, k, d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}


void
matmult_gpu5(int m, int n, int k, double *a, double *b, double *c) {

    double *a_gpu, *b_gpu, *c_gpu;
    int k1, k2;

    hipMalloc((void **) &a_gpu, m * k * sizeof(double));
    hipMalloc((void **) &b_gpu, k * n * sizeof(double));
    hipMalloc((void **) &c_gpu, m * n * sizeof(double));

    hipMemcpy(a_gpu, a, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, k * n * sizeof(double), hipMemcpyHostToDevice);

    k1 = (n-1)/16+1;
    k2 = (m-1)/16+1;
    dim3 dimBlock(16,16,1);
    dim3 dimGrid(k1,k2,1);

    gpu5<<<dimGrid,dimBlock>>>(m, n, k, a_gpu, b_gpu, c_gpu);
    hipDeviceSynchronize();

    hipMemcpy(c, c_gpu, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);

}

void matmult_gpulib(int m, int n, int k, double *h_A, double *h_B, double *h_C)
{
    const double alpha = 1.0, beta = 0.0;
    double *d_A, *d_B, *d_C;

    // Allocate memory on device
    hipMalloc((void **)&d_A, m * k * sizeof(double));
    hipMalloc((void **)&d_B, k * n * sizeof(double));
    hipMalloc((void **)&d_C, m * n * sizeof(double));

    if (d_A == NULL || d_B == NULL || d_C == NULL)
    {
        fprintf(stderr, "memory allocation failed!\n");
        return;
    }

    // Copy data from host to device
    hipMemcpy(d_A, h_A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * n * sizeof(double), hipMemcpyHostToDevice);


    // Create handle for CUBLAS
    hipblasHandle_t handle;
    if(hipblasCreate(&handle)!=HIPBLAS_STATUS_SUCCESS)
    {
        printf("Error initializing CUDA runtime.\n");
        return;
    }

    // Kernel invocation
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B, n, d_A, k, &beta, d_C, n);

    // Destroy handle
    hipblasDestroy(handle);

    checkCudaErrors(hipDeviceSynchronize());

    // Copy result back to host
    hipMemcpy(h_C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
}
