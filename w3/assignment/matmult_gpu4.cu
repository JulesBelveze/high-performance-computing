#include "hip/hip_runtime.h"
// CPU version 4

// nb of elements to compute per thread
#define nb_elt 10

__global__
void gpu4(int m, int n, int k, double *a, double *b, double *c){
    int i,j,l,t;

    i = nb_elt*(blockIdx.y*blockDim.y+threadIdx.y);
    j = blockIdx.x*blockDim.x+threadIdx.x;

    if (i < m-nb_elt && j < n) {
        for (t = 0; t < nb_elt; t++) {
            for (l = 0; l < k; l++) {
                c[(i+t)*n+j] += a[(i+t)*k+l]*b[l*n+j];
            }
        }
    }
    else if (i < m && j < n) {
        for (t = 0; t < nb_elt; t++) {
            if (i+t < m) {
                for (l = 0; l < k; l++) {
                    c[(i+t)*n+j] += a[(i+t)*k+l]*b[l*n+j];
                }
            }
        }
    }
}

void matmult_gpu4(int m, int n, int k, double *a, double *b, double *c){
    double *d_a, *d_b, *d_c;
    int k1, k2;

    // preparing GPU grid and block
    k1 = (n-1)/16+1;
    k2 = (m-1)/16+1;
    dim3 dimBlock(16,16,1);
	dim3 dimGrid(k1,(k2-1)/nb_elts+1,1);

    // allocate space for device copies
    hipMalloc((void **)&d_a, n*k*sizeof(double));
    hipMalloc((void **)&d_b, k*m*sizeof(double));
    hipMalloc((void **)&d_c, m*n*sizeof(double));

    // copying input to device
    hipMemcpy(d_a, a, n*k*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, k*m*sizeof(double), hipMemcpyHostToDevice);

    gpu4<<<dimGrid,dimBlock>>>(m, n, k, d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}
